#include "hip/hip_runtime.h"








 













































typedef char                   sbyte;
typedef unsigned char          byte;
typedef          long long int Int64;
typedef unsigned long long int UInt64;



typedef unsigned short         ushort;
typedef unsigned int           uint;





















 









































extern "C" __device__ float lerp(float n1, float n2, float a)
{
	return n1 + a * (n2 - n1);
}

typedef struct  _Affine2D_struct
{
	float2 xa;
	float2 ya;
	float2 ta;
} Affine2D;

extern "C" __device__ float2 Affine2D_transformVector(const Affine2D* a, float2 v)
{
	return make_float2(a->xa.x*v.x + a->ya.x*v.y + a->ta.x,
	                   a->xa.y*v.x + a->ya.y*v.y + a->ta.y);
}
	
extern "C" __device__ float2 Affine2D_transformVector_cm( Affine2D* a, float2 v)
{
	return make_float2(a->xa.x*v.x + a->ya.x*v.y + a->ta.x,
	                   a->xa.y*v.x + a->ya.y*v.y + a->ta.y);
}
	
extern "C" __device__ void Affine2D_transformPoint(const Affine2D* a, float* x_out, float* y_out, float x, float y)
{
	*x_out = a->xa.x*x + a->ya.x*y + a->ta.x;
	*y_out = a->xa.y*x + a->ya.y*y + a->ta.y;
}

extern "C" __device__ void Affine2D_getInverse(const Affine2D* a, Affine2D* out)
{
	float det = a->xa.x * a->ya.y - a->xa.y * a->ya.x;

	out->xa.x =  a->ya.y / det;
	out->xa.y = -a->xa.y / det;
	out->ya.x = -a->ya.x / det;
	out->ya.y =  a->xa.x / det;
	out->ta.x = (a->ta.y * a->ya.x - a->ta.x * a->ya.y) / det;
	out->ta.y = (a->ta.x * a->xa.y - a->ta.y * a->xa.x) / det;
}


typedef struct  _FractalInfo_struct
{
	uint     branchCount;
	float    brightness;
	float    invGamma;
	float    vibrancy;
	float4   bgColor;
	Affine2D vpsTransform;
	float    reserved0;
	float    reserved1;
} FractalInfo;


typedef struct  _BranchInfo_struct
{
	uint     normWeight;
	float    colorWeight;
	float2   chroma;
	Affine2D preTransform;
	Affine2D postTransform;
} BranchInfo;


typedef struct _IterStatEntry_struct
{
	UInt64 dotCount;
	float  peakDensity;
	float  reserved0;
} IterStatEntry;


typedef struct  _GlobalStatEntry_struct
{
	UInt64 iterCount;
	UInt64 dotCount;
	float  density;
	float  peakDensity;
	float  scaleConstant;
} GlobalStatEntry;


typedef struct  _Dot_struct
{
	float2 pos;
	float2 chroma;
} Dot;


















extern "C" __device__ unsigned int MWC_rand(
	 uint4* x,
	 uint*  c
);

extern "C" __device__ void MWC_seed(
	 uint4* x,
	 uint*  c,
	            uint seed
){
	int j;
	(*x).x = seed * 29943829 - 1;
	(*x).y = (*x).x  * 29943829 - 1;
	(*x).z = (*x).y  * 29943829 - 1;
	(*x).w = (*x).z  * 29943829 - 1;
	*c     = (*x).w  * 29943829 - 1;
	for(j = 0; j < 19; j++)
		MWC_rand(x, c);
}

extern "C" __device__ unsigned int MWC_rand(
	 uint4* x,
	 uint*  c
){
	UInt64 sum =
		((UInt64)2111111111)*(UInt64)(*x).w +
		((UInt64)1492)*(UInt64)(*x).z +
		((UInt64)1776)*(UInt64)(*x).y +
		((UInt64)5115)*(UInt64)(*x).x +
		      (UInt64)(*c);
		      
	(*x).w = (*x).z;
	(*x).z = (*x).y;
	(*x).y = (*x).x;
	(*x).x = (uint)sum;
	*c     = (uint)(sum >> 32);
	return (*x).x;
}

extern "C" __device__ float2 MWC_rand_float2(
	 uint4* x,
	 uint*  c
){
	float2 result;
	uint rnd = MWC_rand(x, c);
	result.x = (float)(rnd&0xFFFF) / 32768.0f - 1.0f;
	result.y = (float)(rnd>>16)   / 32768.0f - 1.0f;
	return result;
}

















































































































































































































extern "C" __device__ float2 vari_linear(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return v;
}

extern "C" __device__ float2 vari_sinusoidal(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		__sinf(v.x),
		__sinf(v.y)
	);
}

extern "C" __device__ float2 vari_spherical(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		v.x / (rsq+(4.7019774E-38f)),
		v.y / (rsq+(4.7019774E-38f))
	);
}

extern "C" __device__ float2 vari_swirl(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float j = __sinf(rsq);
	float k = __cosf(rsq);
	return make_float2(
		j*v.x - k*v.y,
		k*v.x + j*v.y
	);
}
	
extern "C" __device__ float2 vari_horseshoe(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		(v.x*v.x - v.y*v.y)/(r+(4.7019774E-38f)),
		(2.0f*v.x*v.y)/(r+(4.7019774E-38f))
	);
}
	
extern "C" __device__ float2 vari_polar(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		theta / 3.14159265358f,
		r - 1.0f
	);
}
	
extern "C" __device__ float2 vari_handkerchief(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		r * __sinf(theta+r),
		r * __cosf(theta-r)
	);
}
	
extern "C" __device__ float2 vari_heart(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		 r*__sinf(theta*r),
		-r*__cosf(theta*r)
	);
}

extern "C" __device__ float2 vari_disc(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		theta*__sinf(3.14159265358f*r)/3.14159265358f,
		theta*__cosf(3.14159265358f*r)/3.14159265358f
	);
}

extern "C" __device__ float2 vari_spiral(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		(__cosf(theta) + __sinf(r))/(r+(4.7019774E-38f)),
		(__sinf(theta) - __cosf(r))/(r+(4.7019774E-38f))
	);
}

extern "C" __device__ float2 vari_hyperbolic(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		v.y/(rsq+(4.7019774E-38f)),
		v.x
	);
}

extern "C" __device__ float2 vari_diamond(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		v.y / (r+(4.7019774E-38f)) * __cosf(r),
		v.x / (r+(4.7019774E-38f)) * __sinf(r)
	);
}

extern "C" __device__ float2 vari_ex(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float j,k;
	j = __cosf(r)*v.y + __sinf(r)*v.x;
	k = __sinf(r)*v.y + __cosf(r)*v.x;
	return make_float2( j*j*j/(rsq+(4.7019774E-38f)), k*k*k/(rsq+(4.7019774E-38f)) );
}

extern "C" __device__ float2 vari_julia(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float k = 0.5f*theta + 3.14159265358f*(float)(entropy&0x0001);
	return make_float2( __fsqrt_rn(r)*__cosf(k), __fsqrt_rn(r)*__sinf(k) );
}

extern "C" __device__ float2 vari_bent(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2( v.x >= 0 ? v.x : 2.0f*v.x, v.y >= 0 ? v.y : v.y*0.5f);
}

extern "C" __device__ float2 vari_waves(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		v.x + branch->preTransform.ya.x * __sinf(v.y/(branch->preTransform.ta.x*branch->preTransform.ta.x + (4.7019774E-38f))),
		v.y + branch->preTransform.ya.y * __sinf(v.x/(branch->preTransform.ta.y*branch->preTransform.ta.y + (4.7019774E-38f)))
	);
}

extern "C" __device__ float2 vari_fisheye(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float k = 2.0f/(r+1.0f);
	return make_float2(k*v.y, k*v.x);
}

extern "C" __device__ float2 vari_popcorn(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2(
		v.x + branch->preTransform.ta.x*__sinf(__tanf(3.0f*v.y)),
		v.y + branch->preTransform.ta.y*__sinf(__tanf(3.0f*v.x))
	);
}

extern "C" __device__ float2 vari_exponential(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float k = __expf(v.x - 1.0f);
	return make_float2( __sinf(3.14159265358f*v.y)*k, __cosf(3.14159265358f*v.y)*k );
}

extern "C" __device__ float2 vari_power(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float k;
	float nx = v.x/(r+(4.7019774E-38f));
	float ny = v.y/(r+(4.7019774E-38f));
	k = __powf(r,ny);
	return make_float2(nx * k, ny * k);
}

extern "C" __device__ float2 vari_cosine(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float cosh_ty = 0.5f*(__expf(v.y) - __expf(-v.y));
	float sinh_ty = 0.5f*(__expf(v.y) + __expf(-v.y));
	return make_float2( __cosf(3.14159265358f*v.x)*cosh_ty, -1.0f*__sinf(3.14159265358f*v.x)*sinh_ty );
}

extern "C" __device__ float2 vari_eyefish(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float k = 2.0f/(r+1.0f);
	return make_float2( k*v.x, k*v.y );
}

extern "C" __device__ float2 vari_bubble(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float k = 4.0f / (rsq + 4.0f);
	return make_float2( k*v.x, k*v.y );
}

extern "C" __device__ float2 vari_cylinder(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	return make_float2( __sinf(v.x), v.y );
}

extern "C" __device__ float2 vari_noise(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float p1,p2;
	uint rnd = MWC_rand(randX, randC);
	p1 = (float)(rnd>>16) / 65536.0f;
	p2 = 2.0f*3.14159265358f*(float)(rnd&0x0000FFFF) / 65536.0f;
	
	return make_float2( p1*__cosf(p2)*v.x, p1*__sinf(p2)*v.y );
}

extern "C" __device__ float2 vari_blur(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float br,bt;	
	uint rnd = MWC_rand(randX, randC);
	br = (float)(rnd>>16) / 65536.0f;
	bt = 2.0f * 3.14159265358f * (float)(rnd&0x0000FFFF) / 65536.0f;
	
	return make_float2( __cosf(bt)*br, __sinf(bt)*br );
}

extern "C" __device__ float2 vari_gaussian_blur(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float br,bt;
	uint rnd;
	uint sum = 0;
	
	rnd = MWC_rand(randX, randC);
	sum += (rnd&0x0000FFFF) + (rnd>>16);
	rnd = MWC_rand(randX, randC);
	sum += (rnd&0x0000FFFF) + (rnd>>16);
	br = (float)sum / 65536.0f - 2.0f;
	bt = 2.0f * 3.14159265358f * (float)(entropy&0x0000FFFF) / 65536.0f;
	
	return make_float2( __cosf(bt)*br, __sinf(bt)*br );
}

extern "C" __device__ float2 vari_orb9k(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float k = 2.0f/(rsq+1.0f);
	return make_float2( k*v.x, k*v.y );
}

extern "C" __device__ float2 vari_ripple9k(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float k = __sinf(r*1.57079632679f);
	return make_float2( k*v.x, k*v.y );
}

extern "C" __device__ float2 vari_bulge9k(
	float2 v,
	 BranchInfo* branch,
	float theta, float r, float rsq, uint entropy,
	 uint4*      randX,
	 uint*       randC
){
	float k = (r+1.0f)/(r+(4.7019774E-38f));
	return make_float2( k*v.x, k*v.y );
}



extern "C" __device__ uint chooseRandomBranch(
	uint rnd,
	uint branchCount,
	 BranchInfo branches[]
){
	uint i;

	for(i = 0; i < 16; i++)
		if(rnd < branches[i].normWeight)
			return i;
	
	return i;
}








extern "C" __device__ void iterate(
	 float2*   pos,
	 float2*   color,
	            uint      entropy,
	 BranchInfo* branch,
	 float       branchVariWeights[],
	 uint4*      randX,
	 uint*       randC
){
	float2 t;
	float2 vn;
	float2 result;
	float theta, rsq, r;
	result.x = 0.0f;
	result.y = 0.0f;
	
	(*color).x = lerp((*color).x, branch->chroma.x, branch->colorWeight);
	(*color).y = lerp((*color).y, branch->chroma.y, branch->colorWeight);
		
	t = Affine2D_transformVector_cm(&(branch->preTransform), *pos);
	theta = atan2f(t.x,t.y);
	rsq = t.x*t.x + t.y*t.y;
	r = __fsqrt_rn(rsq);
	
	if(branchVariWeights[0] > 0.0f){ vn = vari_linear(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[0]*vn.x; result.y += branchVariWeights[0]*vn.y; }
	if(branchVariWeights[1] > 0.0f){ vn = vari_sinusoidal(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[1]*vn.x; result.y += branchVariWeights[1]*vn.y; }
	if(branchVariWeights[2] > 0.0f){ vn = vari_spherical(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[2]*vn.x; result.y += branchVariWeights[2]*vn.y; }
	if(branchVariWeights[3] > 0.0f){ vn = vari_swirl(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[3]*vn.x; result.y += branchVariWeights[3]*vn.y; }
	if(branchVariWeights[4] > 0.0f){ vn = vari_horseshoe(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[4]*vn.x; result.y += branchVariWeights[4]*vn.y; }
	if(branchVariWeights[5] > 0.0f){ vn = vari_polar(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[5]*vn.x; result.y += branchVariWeights[5]*vn.y; }
	if(branchVariWeights[6] > 0.0f){ vn = vari_handkerchief(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[6]*vn.x; result.y += branchVariWeights[6]*vn.y; }
	if(branchVariWeights[7] > 0.0f){ vn = vari_heart(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[7]*vn.x; result.y += branchVariWeights[7]*vn.y; }
	if(branchVariWeights[8] > 0.0f){ vn = vari_disc(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[8]*vn.x; result.y += branchVariWeights[8]*vn.y; }
	if(branchVariWeights[9] > 0.0f){ vn = vari_spiral(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[9]*vn.x; result.y += branchVariWeights[9]*vn.y; }
	if(branchVariWeights[10] > 0.0f){ vn = vari_hyperbolic(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[10]*vn.x; result.y += branchVariWeights[10]*vn.y; }
	if(branchVariWeights[11] > 0.0f){ vn = vari_diamond(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[11]*vn.x; result.y += branchVariWeights[11]*vn.y; }
	if(branchVariWeights[12] > 0.0f){ vn = vari_ex(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[12]*vn.x; result.y += branchVariWeights[12]*vn.y; }
	if(branchVariWeights[13] > 0.0f){ vn = vari_julia(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[13]*vn.x; result.y += branchVariWeights[13]*vn.y; }
	if(branchVariWeights[14] > 0.0f){ vn = vari_bent(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[14]*vn.x; result.y += branchVariWeights[14]*vn.y; }
	if(branchVariWeights[15] > 0.0f){ vn = vari_waves(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[15]*vn.x; result.y += branchVariWeights[15]*vn.y; }
	if(branchVariWeights[16] > 0.0f){ vn = vari_fisheye(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[16]*vn.x; result.y += branchVariWeights[16]*vn.y; }
	if(branchVariWeights[17] > 0.0f){ vn = vari_popcorn(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[17]*vn.x; result.y += branchVariWeights[17]*vn.y; }
	if(branchVariWeights[18] > 0.0f){ vn = vari_exponential(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[18]*vn.x; result.y += branchVariWeights[18]*vn.y; }
	if(branchVariWeights[19] > 0.0f){ vn = vari_power(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[19]*vn.x; result.y += branchVariWeights[19]*vn.y; }
	if(branchVariWeights[20] > 0.0f){ vn = vari_cosine(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[20]*vn.x; result.y += branchVariWeights[20]*vn.y; }
	if(branchVariWeights[21] > 0.0f){ vn = vari_eyefish(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[21]*vn.x; result.y += branchVariWeights[21]*vn.y; }
	if(branchVariWeights[22] > 0.0f){ vn = vari_bubble(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[22]*vn.x; result.y += branchVariWeights[22]*vn.y; }
	if(branchVariWeights[23] > 0.0f){ vn = vari_cylinder(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[23]*vn.x; result.y += branchVariWeights[23]*vn.y; }
	if(branchVariWeights[24] > 0.0f){ vn = vari_noise(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[24]*vn.x; result.y += branchVariWeights[24]*vn.y; }
	if(branchVariWeights[25] > 0.0f){ vn = vari_blur(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[25]*vn.x; result.y += branchVariWeights[25]*vn.y; }
	if(branchVariWeights[26] > 0.0f){ vn = vari_gaussian_blur(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[26]*vn.x; result.y += branchVariWeights[26]*vn.y; }
	if(branchVariWeights[27] > 0.0f){ vn = vari_orb9k(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[27]*vn.x; result.y += branchVariWeights[27]*vn.y; }
	if(branchVariWeights[28] > 0.0f){ vn = vari_ripple9k(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[28]*vn.x; result.y += branchVariWeights[28]*vn.y; }
	if(branchVariWeights[29] > 0.0f){ vn = vari_bulge9k(t, branch, theta, r, rsq, entropy, randX, randC); result.x += branchVariWeights[29]*vn.x; result.y += branchVariWeights[29]*vn.y; }
	
	*pos = Affine2D_transformVector_cm(&(branch->postTransform), result);
}













__constant__    FractalInfo      fractalInfo[1];
__constant__    BranchInfo       branchInfo[16];
__constant__    float            variWeightBuffer[16*48];

texture<float4, 2, hipReadModeElementType> paletteTex;


extern "C" __global__ void init_iterators_kernel(
	 uint4 entropyXBuffer[],
	 uint  entropyCBuffer[],
	 uint  entropySeedBuffer[]
){
	__shared__ uint4 randXBuffer   [64];
	__shared__ uint  randCBuffer   [64];
	__shared__ uint  randSeedBuffer[64];
	
	uint lid = threadIdx.x;
	uint gid = (blockIdx.x*blockDim.x + threadIdx.x);
	
	randXBuffer[lid]    = make_uint4(0,0,0,0);
	randCBuffer[lid]    = 0;
	randSeedBuffer[lid] = entropySeedBuffer[gid];
	
	MWC_seed(randXBuffer+lid, randCBuffer+lid, randSeedBuffer[lid]);
	
	entropyXBuffer[gid] = randXBuffer[lid];
	entropyCBuffer[gid] = randCBuffer[lid];
}

extern "C" __global__ void reset_iterators_kernel(
	             uint             xRes,
	             uint             yRes,





	   float2           iterPosStateBuffer[],
	   float2           iterColorStateBuffer[],
	   IterStatEntry    iterStatBuffer[],
	   GlobalStatEntry* globalStatBuffer,
	   uint4            entropyXBuffer[],
	   uint             entropyCBuffer[],
	   uint             entropySeedBuffer[]
){
	int iter;
	uint bi;
	float2 pos;
	float2 color;
	uint rnd;

	__shared__ uint4 randXBuffer   [64];
	__shared__ uint  randCBuffer   [64];
	__shared__ uint  randSeedBuffer[64];
	
	uint lid = threadIdx.x;
	uint gid = (blockIdx.x*blockDim.x + threadIdx.x);
	
	randXBuffer[lid]    = make_uint4(0,0,0,0);
	randCBuffer[lid]    = 0;
	randSeedBuffer[lid] = entropySeedBuffer[gid];
	
	MWC_seed(randXBuffer+lid, randCBuffer+lid, randSeedBuffer[lid]);
	
	pos = MWC_rand_float2(randXBuffer+lid, randCBuffer+lid);
	
	
	
	
	color = make_float2(0.5f, 0.5f);
	
	for(iter = 0; iter < 32; iter++)
	{
		rnd = MWC_rand(randXBuffer+lid, randCBuffer+lid);
		bi = chooseRandomBranch(rnd & 0x0000FFFF, fractalInfo->branchCount, branchInfo);                
		iterate(&pos, &color, (rnd>>16), branchInfo+bi, variWeightBuffer+(bi*48), randXBuffer+lid, randCBuffer+lid); 
	}
	
	iterPosStateBuffer[gid] = pos;
	iterColorStateBuffer[gid] = color;
	iterStatBuffer[gid].dotCount = 0;
	iterStatBuffer[gid].peakDensity = 0.0f;
	
	if(gid == 0) 
	{
		globalStatBuffer->iterCount =     0;
		globalStatBuffer->dotCount =      0;
		globalStatBuffer->density  =      0.0f;
		globalStatBuffer->peakDensity =   0.0f;
		globalStatBuffer->scaleConstant = 0.0f;
	}
}

extern "C" __global__ void reset_output_kernel(
	int xRes,
	int yRes,
	 float4  accumBuffer[],
	 uint    outputBuffer[]
)
{
	int x = (blockIdx.x*blockDim.x + threadIdx.x);
	int y = (blockIdx.y*blockDim.y + threadIdx.y);

	if(x < xRes && y < yRes)
	{
		(*((accumBuffer + 4*((((y))*xRes) + ((x))) + 2*((0)) + ((0))))) = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		(*((accumBuffer + 4*((((y))*xRes) + ((x))) + 2*((1)) + ((0))))) = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		(*((accumBuffer + 4*((((y))*xRes) + ((x))) + 2*((0)) + ((1))))) = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		(*((accumBuffer + 4*((((y))*xRes) + ((x))) + 2*((1)) + ((1))))) = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		
		(*((outputBuffer + (((y))*xRes) + ((x))))) = 0x00000000;
	}
}














extern "C" __global__ void iterate_kernel(
	             uint             xRes,
	             uint             yRes,





	   float2           iterPosStateBuffer[],
	   float2           iterColorStateBuffer[],
	   IterStatEntry    iterStatBuffer[],
	   GlobalStatEntry* globalStatBuffer,
	   uint4            entropyXBuffer[],
	   uint             entropyCBuffer[],
	   float4           accumBuffer[],










	             uint             iterCount
){
	float2 pos;
	float2 color;
	float4 sample = make_float4(1.0f,1.0f,1.0f,1.0f);
	
	float4 mem;
	uint bi;
	float2 screenPos;
	UInt64 dotCount;
	float peakDensity;
	int xa,ya;
	int x,y;
	uint iter;
	uint rnd;
		
	__shared__ uint4 randXBuffer   [64];
	__shared__ uint  randCBuffer   [64];
	
	uint lid = threadIdx.x;
	uint gid = (blockIdx.x*blockDim.x + threadIdx.x);

	pos =       iterPosStateBuffer[gid];
	color =     iterColorStateBuffer[gid];
	dotCount =  iterStatBuffer[gid].dotCount;
	peakDensity = iterStatBuffer[gid].peakDensity;
	randXBuffer[lid] = entropyXBuffer[gid];
	randCBuffer[lid] = entropyCBuffer[gid];
	
	
	for(iter = 0; iter < iterCount; iter++)
	{
		rnd = MWC_rand(randXBuffer+lid, randCBuffer+lid);
		bi = chooseRandomBranch(rnd & 0x0000FFFF, fractalInfo->branchCount, branchInfo);                  
		
		iterate(&pos, &color, (rnd>>16), branchInfo+bi, variWeightBuffer+(bi*48), randXBuffer+lid, randCBuffer+lid); 
		
		screenPos = Affine2D_transformVector_cm(&(fractalInfo->vpsTransform), pos);
		
		xa = (int)(2.0f*screenPos.x);
		ya = (int)(2.0f*screenPos.y);
		x  = xa >> 1;
		y  = ya >> 1;
		
		if(x >= 0 && x < xRes && y >= 0 && y < yRes)
		{








			sample = tex2D(paletteTex, color.x, color.y);

			
			
			
			mem = (*((accumBuffer + 4*((((y))*xRes) + ((x))) + 2*(((ya&1))) + (((xa&1))))));
			mem.x += sample.x;
			mem.y += sample.y;
			mem.z += sample.z;
			mem.w += 1.0f;
			(*((accumBuffer + 4*((((y))*xRes) + ((x))) + 2*(((ya&1))) + (((xa&1)))))) = mem;
			
			dotCount++;
			peakDensity = fmaxf(peakDensity,mem.w);
		}
	}
	
	
	iterPosStateBuffer[gid]   = pos;
	iterColorStateBuffer[gid] = color;
	iterStatBuffer[gid].dotCount = dotCount;
	iterStatBuffer[gid].peakDensity = peakDensity;
	entropyXBuffer[gid] = randXBuffer[lid];
	entropyCBuffer[gid] = randCBuffer[lid];
	
	if(gid == 0)
	{
		globalStatBuffer->iterCount += (UInt64)((gridDim.x*blockDim.x) * iterCount);
	}
}


extern "C" __global__ void update_stats_kernel(
                 uint             xRes,
                 uint             yRes,



                 uint             iteratorCount,
	   IterStatEntry    iterStatBuffer[],
	   GlobalStatEntry* globalStatBuffer 
){
	UInt64 totalIterationCount = 0;
	UInt64 totalDotCount = 0;
	float peakDensity = 0.0f;
	int i;
	float totalSubPixels, density, invPixArea, scaleConstant;
	
	if((blockIdx.x*blockDim.x + threadIdx.x) == 0)
	{
		for(i = 0; i < iteratorCount; i++)
		{
			totalDotCount += iterStatBuffer[i].dotCount;
			peakDensity = fmaxf(peakDensity,iterStatBuffer[i].peakDensity);
		}
		totalIterationCount = globalStatBuffer->iterCount;
		totalSubPixels = (float)(xRes*yRes*(2*2));
		density = (float)totalDotCount / totalSubPixels;
		invPixArea = fabsf((fractalInfo->vpsTransform.xa.x)*(fractalInfo->vpsTransform.ya.y) - (fractalInfo->vpsTransform.xa.y)*(fractalInfo->vpsTransform.ya.x));
		scaleConstant = (64.0f/1.0f)*(invPixArea*(float)(2*2))/(float)totalIterationCount;
		
		globalStatBuffer->dotCount = totalDotCount;
		globalStatBuffer->density = fmaxf(density,(4.7019774E-38f));
		globalStatBuffer->peakDensity = fmaxf(peakDensity,(4.7019774E-38f));
		globalStatBuffer->scaleConstant = fmaxf(scaleConstant,(4.7019774E-38f));
	}
}

extern "C" __device__ float4 tonemap( FractalInfo* fractal, float4 rawPix, float scaleConstant)
{
	float z, ka, gammaFactor;
	float4 logPix;
	float4 result;        
	
	if(rawPix.w <= 0.5) 
		return make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	
	logPix.w = (1.0f/2.0f) * fractal->brightness * __log10f(1.0f+rawPix.w*scaleConstant);
	ka = logPix.w / rawPix.w;
	
	logPix.x = ka * rawPix.x;
	logPix.y = ka * rawPix.y;
	logPix.z = ka * rawPix.z;
	
	z = __powf(logPix.w,fractal->invGamma);
	gammaFactor = z / logPix.w;
	
	result.x = __saturatef(lerp(__powf(logPix.x,fractal->invGamma), gammaFactor*logPix.x, fractal->vibrancy));
	result.y = __saturatef(lerp(__powf(logPix.y,fractal->invGamma), gammaFactor*logPix.y, fractal->vibrancy));
	result.z = __saturatef(lerp(__powf(logPix.z,fractal->invGamma), gammaFactor*logPix.z, fractal->vibrancy));
	result.w = __saturatef(z);
	
	return result;
}

extern "C" __global__ void update_output_kernel(
	           uint             xRes,
	           uint             yRes,



	 GlobalStatEntry* globalStatBuffer,
	 float4           accumBuffer[],
	 uint             outputBuffer[]
){
	uint4 iPix;
	float4 pix,acc;
	float scaleConstant;
	
	int x = (blockIdx.x*blockDim.x + threadIdx.x);
	int y = (blockIdx.y*blockDim.y + threadIdx.y);
	
	acc = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		
	if(x < xRes && y < yRes)
	{
		scaleConstant = globalStatBuffer->scaleConstant;
		
		pix = tonemap( fractalInfo, (*((accumBuffer + 4*((((y))*xRes) + ((x))) + 2*((0)) + ((0))))), scaleConstant);
		acc.x += pix.w*pix.x;
		acc.y += pix.w*pix.y;
		acc.z += pix.w*pix.z;
		acc.w += pix.w;
			
		pix = tonemap( fractalInfo, (*((accumBuffer + 4*((((y))*xRes) + ((x))) + 2*((1)) + ((0))))), scaleConstant);
		acc.x += pix.w*pix.x;
		acc.y += pix.w*pix.y;
		acc.z += pix.w*pix.z;
		acc.w += pix.w;
		
		pix = tonemap( fractalInfo, (*((accumBuffer + 4*((((y))*xRes) + ((x))) + 2*((0)) + ((1))))), scaleConstant);
		acc.x += pix.w*pix.x;
		acc.y += pix.w*pix.y;
		acc.z += pix.w*pix.z;
		acc.w += pix.w;
		
		pix = tonemap( fractalInfo, (*((accumBuffer + 4*((((y))*xRes) + ((x))) + 2*((1)) + ((1))))), scaleConstant);
		acc.x += pix.w*pix.x;
		acc.y += pix.w*pix.y;
		acc.z += pix.w*pix.z;
		acc.w += pix.w;
		
		if(acc.w < (1.0f/256.0f))
		{
			iPix = make_uint4(0,0,0,0);
		}
		else
		{
			acc.x /= acc.w;
			acc.y /= acc.w;
			acc.z /= acc.w;
			acc.w *= 0.25f;
					
			iPix.x = (uint)(255.0f*acc.x) & 0xFF;
			iPix.y = (uint)(255.0f*acc.y) & 0xFF;
			iPix.z = (uint)(255.0f*acc.z) & 0xFF;
			iPix.w = (uint)(255.0f*acc.w) & 0xFF;
		}
				
		(*((outputBuffer + (((y))*xRes) + ((x))))) = iPix.w << 24 | iPix.z << 16 | iPix.y << 8 | iPix.x;
	}
}




































































































