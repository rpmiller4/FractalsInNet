#include "hip/hip_runtime.h"
/*
    Fractron 9000
    Copyright (C) 2009 Michael J. Thiesen
	http://fractron9000.sourceforge.net
	mike@thiesen.us

    This program is free software; you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation; either version 2 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program; if not, write to the Free Software
    Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.
*/

#include "cuda_interop.cuh"
#include "RandomMWC.cuh"
#include "Affine2D.cuh"

#define IterBlockSize 128
#define IteratorCount (IterBlockSize*iterBlockCount)
#define AALevel 2
#define SubPixelCount (AALevel*AALevel)
#define RasterBlockSize 8
#define MaxBranches 16
#define MaxFactors 48

#define WarmupIterationCount 32

#define Tone_C1 (1.0f/2.0f)
#define Tone_C2 (64.0f/1.0f)
#define PIf 3.14159265358f
#define PIo2f 1.57079632679f
#define InvPIo2f 0.636619772368f
//#define Epsilon 0.00000095367431640625f
#define Epsilon (4.7019774E-38f)


__constant__ int xRes;
__constant__ int yRes;
__constant__ int iterBlockCount;

__constant__ DeviceBuffer2D<float2> iterPosStateBuffer;
__constant__ DeviceBuffer2D<float4> iterColorStateBuffer;
__constant__ DeviceBuffer2D<uint4>  entropyXBuffer;
__constant__ DeviceBuffer2D<uint>   entropyCBuffer;
__constant__ DeviceBuffer2D<uint>   entropySeedBuffer;
__constant__ DeviceBuffer2D<UInt64> dotCountBuffer;
__constant__ DeviceBuffer2D<float>  peakDensityBuffer;

__constant__ UInt64* totalIterCountMem;
__constant__ UInt64* totalDotCountMem;
__constant__ float* densityMem;
__constant__ float* peakDensityMem;
__constant__ float* scaleConstantMem;

__constant__ DeviceBuffer2D<float4> accumBuffer;

__constant__ Affine2D vpsTransform; //transforms a point from world space to screen space

__constant__ float brightness;
__constant__ float invGamma;
__constant__ float vibrancy;
__constant__ float4 bgColor;

__constant__ uint     branchCount;
__constant__ uint     branchNormWeights[MaxBranches];
__constant__ float    branchColorWeights[MaxBranches];
__constant__ Affine2D branchPreTransforms[MaxBranches];
__constant__ Affine2D branchPostTransforms[MaxBranches];
__constant__ float    branchLumas[MaxBranches];
__constant__ float2   branchChromas[MaxBranches];
__constant__ float    branchFactors[MaxBranches*MaxFactors];

texture<float4, 2, hipReadModeElementType> paletteTex;

__device__ float lerp( float n1, float n2, float a )
{
	return n1 + a * ( n2 - n1 );
}


__device__ void iterate(float2* pos, float4* color, uint branch, uint entropy, uint4* randXBuffer, uint* randCBuffer)
{
	#define AccumResult(i) result.x += factors[i]*nx; result.y += factors[i]*ny
	float tx,ty;
	float nx,ny;
	float* factors = branchFactors + branch*MaxFactors;
	
	float2 result;
	result.x = 0.0f;
	result.y = 0.0f;
	
	color->x = lerp(color->x, branchChromas[branch].x, branchColorWeights[branch]);
	color->y = lerp(color->y, branchChromas[branch].y, branchColorWeights[branch]);
	color->z = lerp(color->z, branchLumas[branch]    , branchColorWeights[branch]);
	
	Affine2D pre;
	Affine2D post;
	
	pre = branchPreTransforms[branch];
	post = branchPostTransforms[branch];
	
	pre.transformPoint(&tx, &ty, pos->x, pos->y);
	
	float theta = atan2f(tx,ty); 
	float rsq = tx*tx + ty*ty;
	float r = __fsqrt_ru(rsq);
	
	//Linear
	if(factors[0] != 0.0f)
	{
		nx = tx;
		ny = ty;
		AccumResult(0);
	}
	
	//Apophysis Sinusoidal
	if(factors[1] != 0.0f)
	{
		nx = __sinf(tx);
		ny = __sinf(ty);
		AccumResult(1);
	}
	
	//Apophysis Spherical
	if(factors[2] != 0.0f)
	{
		nx = tx / (rsq+Epsilon);
		ny = ty / (rsq+Epsilon);
		AccumResult(2);
	}

	//Apophysis Swirl
	if(factors[3] != 0.0f)
	{
		float j,k;
		__sincosf(rsq, &j, &k);
		nx = j*tx - k*ty;
		ny = k*tx + j*ty;
		AccumResult(3);
	}
	
	//Apophysis Horseshoe
	if(factors[4] != 0.0f)
	{
		nx = (tx*tx - ty*ty)/(r+Epsilon);
		ny = (2.0f*tx*ty)/(r+Epsilon);
		AccumResult(4);
	}
	
	//Apophysis Polar
	if(factors[5] != 0.0f)
	{
		nx = theta / PIf;
		ny = r - 1.0f;
		AccumResult(5);
	}
	
	//Apophysis Handkerchief
	if(factors[6] != 0.0f)
	{
		nx = r * __sinf(theta+r);
		ny = r * __cosf(theta-r);
		AccumResult(6);
	}
	
	//Apophysis Heart
	if(factors[7] != 0.0f)
	{
		nx =  r*__sinf(theta*r);
		ny = -r*__cosf(theta*r);
		AccumResult(7);
	}
	
	//Apophysis Disc
	if(factors[8] != 0.0f)
	{
		nx =  theta*__sinf(PIf*r)/PIf;
		ny =  theta*__cosf(PIf*r)/PIf;
		AccumResult(8);
	}
	
	//Apophysis Spiral
	if(factors[9] != 0.0f)
	{
		nx =  (__cosf(theta) + __sinf(r))/(r+Epsilon);
		ny =  (__sinf(theta) - __cosf(r))/(r+Epsilon);
		AccumResult(9);
	}
	
	//Apophysis Hyperbolic
	if(factors[10] != 0.0f)
	{
		nx =  ty/(rsq+Epsilon);
		ny =  tx;
		AccumResult(10);
	}
	
	//Apophysis Diamond
	if(factors[11] != 0.0f)
	{
		nx =  ty / (r+Epsilon) * __cosf(r);
		ny =  tx / (r+Epsilon) * __sinf(r);
		AccumResult(11);
	}
	
	//Apophysis Ex
	if(factors[12] != 0.0f)
	{
		float j,k;
		j = __cosf(r)*ty + __sinf(r)*tx;
		nx =  j*j*j/(rsq+Epsilon);
		k = __sinf(r)*ty + __cosf(r)*tx;
		ny =  k*k*k/(rsq+Epsilon);
		AccumResult(12);
	}
	
	//Apophysis Julia
	if(factors[13] != 0.0f)
	{
		float j,k;
		__sincosf(0.5f*theta + PIf*(float)(entropy&0x0001), &j, &k);
		nx = sqrtf(r) * k;
		ny = sqrtf(r) * j;
		AccumResult(13);
	}
	
	//Apophysis Bent
	if(factors[14] != 0.0f)
	{
		nx = tx >= 0 ? tx : 2*tx;
		ny = ty >= 0 ? ty : ty*0.5f;
		AccumResult(14);
	}
	
	//Apophysis Waves
	if(factors[15] != 0.0f)
	{
		nx = tx + pre.ya.x * __sinf(ty/(pre.ta.x*pre.ta.x + Epsilon));
		ny = ty + pre.ya.y * __sinf(tx/(pre.ta.y*pre.ta.y + Epsilon));
		AccumResult(15);
	}
	
	//Apophysis Fisheye
	if(factors[16] != 0.0f)
	{
		float k;
		k = 2.0f/(r+1.0f);
		nx = k*ty;
		ny = k*tx;
		AccumResult(16);
	}
	
	//Apophysis Popcorn
	if(factors[17] != 0.0f)
	{
		nx = tx + pre.ta.x*__sinf(__tanf(3.0f*ty));
		ny = ty + pre.ta.y*__sinf(__tanf(3.0f*tx));
		AccumResult(17);
	}
	
	//Apophysis Exponential
	if(factors[18] != 0.0f)
	{
		float k;
		k = __expf(tx - 1.0f);
		__sincosf(PIf*ty, &nx, &ny);
		nx *= k;
		ny *= k;
		AccumResult(18);
	}
	
	//Apophysis Power
	if(factors[19] != 0.0f)
	{
		float k;
		nx = tx/(r+Epsilon);
		ny = ty/(r+Epsilon);
		k = __powf(r, ny);
		nx *= k;
		ny *= k;
		AccumResult(19);
	}
	
	//Apophysis Cosine
	if(factors[20] != 0.0f)
	{
		float sinh_ty, cosh_ty;
		cosh_ty = 0.5f*(__expf(ty) - __expf(-ty));
		sinh_ty = 0.5f*(__expf(ty) + __expf(-ty));
		nx =       __cosf(PIf*tx)*cosh_ty;
		ny = -1.0f*__sinf(PIf*tx)*sinh_ty;
		AccumResult(20);
	}
	
	//Apophysis Eyefish
	if(factors[21] != 0.0f)
	{
		float k;
		k = 2.0f/(r+1.0f);
		nx = k*tx;
		ny = k*ty;
		AccumResult(21);
	}
	
	//Apophysis Bubble
	if(factors[22] != 0.0f)
	{
		float k;
		k = 4.0f / (rsq + 4.0f);
		nx = k*tx;
		ny = k*ty;
		AccumResult(22);
	}
	
	//Apophysis Cylinder
	if(factors[23] != 0.0f)
	{
		nx = __sinf(tx);
		ny = ty;
		AccumResult(23);
	}
	
	//Apophysis Noise
	if(factors[24] != 0.0f)
	{
		float p1,p2;
		
		uint rnd = MWC_rand(randXBuffer, randCBuffer, threadIdx.x);
		p1 = (float)(rnd>>16) / 65536.0f;
		p2 = 2.0f*PIf*(float)(rnd&0x0000FFFF) / 65536.0f;
		
		__sincosf(p2, &ny, &nx);
		nx *= p1*tx;
		ny *= p1*ty;

		AccumResult(24);
	}
	
	//Apophysis Blur
	if(factors[25] != 0.0f)
	{
		float br,bt;
		
		uint rnd = MWC_rand(randXBuffer, randCBuffer, threadIdx.x);
		br = (float)(rnd>>16) / 65536.0f;
		bt = 2.0f * PIf * (float)(rnd&0x0000FFFF) / 65536.0f;
		
		__sincosf(bt, &ny, &nx);
		nx *= br;
		ny *= br;

		AccumResult(25);
	}
	
	//Apophysis Gaussian Blur
	if(factors[26] != 0.0f)
	{
		float br,bt;
		uint rnd;
		uint sum = 0;
		
		rnd = MWC_rand(randXBuffer, randCBuffer, threadIdx.x);
		sum += (rnd&0x0000FFFF) + (rnd>>16);
		rnd = MWC_rand(randXBuffer, randCBuffer, threadIdx.x);
		sum += (rnd&0x0000FFFF) + (rnd>>16);
		br = (float)sum / 65536.0f - 2.0f;
		bt = 2.0f * PIf * (float)(entropy&0x0000FFFF) / 65536.0f;
		
		__sincosf(bt, &ny, &nx);
		nx *= br;
		ny *= br;

		AccumResult(26);
	}
	
	//Fractron Orb
	if(factors[27] != 0.0f)
	{
		float k;
		k = 2.0f/(rsq+1.0f);
		nx = k*tx;
		ny = k*ty;
		AccumResult(27);
	}
	
	//Fractron Ripple
	if(factors[28] != 0.0f)
	{
		float k;
		k = __sinf(r*PIo2f);
		nx = k*tx;
		ny = k*ty;
		AccumResult(28);
	}
	
	//Fractron Bulge
	if(factors[29] != 0.0f)
	{
		float k;
		k = (r+1.0f)/(r+Epsilon);
		
		nx = k*tx;
		ny = k*ty;
		AccumResult(29);
	}
	
	post.transformPoint(&(result.x), &(result.y), result.x, result.y);
	
	#undef AccumResult
	
	*pos = result;
}

//chooses a branch index randomly based on the branch weights
__device__ uint chooseRandomBranch(uint rnd)
{
	uint branch = 0;

	#pragma unroll
	for(int i = 0; i < MaxBranches; i++)
		if(rnd >= branchNormWeights[i])
			branch++;
	
	//while(branch < branchCount-1 && rnd >= branchNormWeights[branch])
	//	branch++;
	return branch;
}

extern "C"
__global__ void init_iterators_kernel()
{
	#define IterIdx (blockDim.x*blockIdx.x + threadIdx.x)
	
	__shared__ uint4 randXBuffer   [IterBlockSize];
	__shared__ uint  randCBuffer   [IterBlockSize];
	__shared__ uint  randSeedBuffer[IterBlockSize];
	
	randXBuffer   [threadIdx.x] = make_uint4(0,0,0,0);
	randCBuffer   [threadIdx.x] = 0;
	randSeedBuffer[threadIdx.x] = entropySeedBuffer[blockIdx.x][threadIdx.x];
	
	MWC_seed(randXBuffer, randCBuffer, threadIdx.x, randSeedBuffer);
	
	entropyXBuffer[blockIdx.x][threadIdx.x] = randXBuffer[threadIdx.x];
	entropyCBuffer[blockIdx.x][threadIdx.x] = randCBuffer[threadIdx.x];
}

extern "C"
__global__ void reset_iterators_kernel(DeviceBuffer2D<uint> output)
{
	int iter;
	uint branch;
	float rndf;
	float2 pos;
	float4 color;
	
	__shared__ uint4 randXBuffer   [IterBlockSize];
	__shared__ uint  randCBuffer   [IterBlockSize];
	__shared__ uint  randSeedBuffer[IterBlockSize];
	randXBuffer   [threadIdx.x] = entropyXBuffer   [blockIdx.x][threadIdx.x];
	randCBuffer   [threadIdx.x] = entropyCBuffer   [blockIdx.x][threadIdx.x];
	randSeedBuffer[threadIdx.x] = entropySeedBuffer[blockIdx.x][threadIdx.x];
	
	MWC_seed(randXBuffer, randCBuffer, threadIdx.x, randSeedBuffer);


	rndf = (float)(MWC_rand(randXBuffer, randCBuffer, threadIdx.x) & 0x000FFFFF) / 1048575.0f;
	pos.x = 2.0f*rndf - 1.0f;
	rndf = (float)(MWC_rand(randXBuffer, randCBuffer, threadIdx.x) & 0x000FFFFF) / 1048575.0f;
	pos.y = 2.0f*rndf - 1.0f;
	
	color.x = 0.5f;
	color.y = 0.5f;
	color.z = 0.5f;
	color.w = 0.0f;
	
	for(iter = 0; iter < WarmupIterationCount; iter++)
	{
		uint rnd = MWC_rand(randXBuffer, randCBuffer, threadIdx.x);
		
		branch = chooseRandomBranch(rnd & 0x0000FFFF); //the low entropy bits are for branch selection
		
		iterate(&pos, &color, branch, (rnd>>16), randXBuffer, randCBuffer);  //the extra entropy bits are for variations
	}

	iterPosStateBuffer  [blockIdx.x][threadIdx.x] = pos;
	iterColorStateBuffer[blockIdx.x][threadIdx.x] = color;
	entropyXBuffer   [blockIdx.x][threadIdx.x] = randXBuffer[threadIdx.x];
	entropyCBuffer   [blockIdx.x][threadIdx.x] = randCBuffer[threadIdx.x];
	dotCountBuffer   [blockIdx.x][threadIdx.x] = 0;
	peakDensityBuffer[blockIdx.x][threadIdx.x] = 0.0f;
	
	if(threadIdx.x == 0 && blockIdx.x == 0) //the first thread will update the total iteration count
	{
		*totalIterCountMem = 0;
	}
}

extern "C"
__global__ void reset_output_kernel(DeviceBuffer2D<uint> glOutputBuffer)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x >= xRes || y >= yRes) return;
	
	accumBuffer[y][4*x+0] = make_float4(0.0f,0.0f,0.0f,0.0f);
	accumBuffer[y][4*x+1] = make_float4(0.0f,0.0f,0.0f,0.0f);
	accumBuffer[y][4*x+2] = make_float4(0.0f,0.0f,0.0f,0.0f);
	accumBuffer[y][4*x+3] = make_float4(0.0f,0.0f,0.0f,0.0f);
	
	glOutputBuffer[y][x] = 0xFF000000;
}

extern "C"
__global__ void iterate_kernel(uint iterCount)
{

	//float r;
	float2 pos;
	float4 color;
	float4 sample;
	
	float4 mem;
	float2 screenPos;
	uint branch;
	UInt64 dotCount;
	float peakCount;
	int xa,ya;
	int x;
	int y;
	int si;
	uint iter;
	
	dotCount = dotCountBuffer[blockIdx.x][threadIdx.x];
	peakCount = peakDensityBuffer[blockIdx.x][threadIdx.x];
	
	__shared__ uint4 randXBuffer[IterBlockSize];
	__shared__ uint  randCBuffer[IterBlockSize];
	randXBuffer[threadIdx.x] = entropyXBuffer[blockIdx.x][threadIdx.x];
	randCBuffer[threadIdx.x] = entropyCBuffer[blockIdx.x][threadIdx.x];
	//MWC_seed(randXBuffer, randCBuffer, threadIdx.x, seed + idx);

	pos =   iterPosStateBuffer[blockIdx.x][threadIdx.x];
	color = iterColorStateBuffer[blockIdx.x][threadIdx.x];

	for(iter = 0; iter < iterCount; iter++)
	{
		uint rnd = MWC_rand(randXBuffer, randCBuffer, threadIdx.x);

		branch = chooseRandomBranch(rnd & 0x0000FFFF); //the low entropy bits are for branch selection

		iterate(&pos, &color, branch, (rnd>>16), randXBuffer, randCBuffer); //the extra entropy is for variations
		
		vpsTransform.transformPoint(&screenPos.x, &screenPos.y, pos.x, pos.y);
		xa = 2.0f*screenPos.x;
		ya = 2.0f*screenPos.y;
		x = xa >> 1;
		y = ya >> 1;
		
		if(x >= 0 && x < xRes && y >= 0 && y < yRes)
		{
			si = ((ya&1)<<1) | (xa&1); //calc subpixel index
			
			//sample the color from the palette
			sample = tex2D(paletteTex, color.x, color.y);
			
			//accumulate the histogram buffer
			//this is not actually thread safe, but hopefully it wont screw up the counts
			//enough to trash the image
			mem = accumBuffer[y][4*x+si];
			mem.x += sample.x * color.z;
			mem.y += sample.y * color.z;
			mem.z += sample.z * color.z;
			mem.w += 1.0f;
			accumBuffer[y][4*x+si] = mem;
						
			dotCount++;
			peakCount = fmaxf(peakCount, mem.w);
		}
	}

	iterPosStateBuffer  [blockIdx.x][threadIdx.x] = pos;
	iterColorStateBuffer[blockIdx.x][threadIdx.x] = color;
	entropyXBuffer   [blockIdx.x][threadIdx.x] = randXBuffer[threadIdx.x];
	entropyCBuffer   [blockIdx.x][threadIdx.x] = randCBuffer[threadIdx.x];
	dotCountBuffer   [blockIdx.x][threadIdx.x] = dotCount;
	peakDensityBuffer[blockIdx.x][threadIdx.x] = peakCount;
	
	if(threadIdx.x == 0 && blockIdx.x == 0) //the first thread will update the total iteration count
	{
		*totalIterCountMem += (UInt64)(IterBlockSize*iterBlockCount*iterCount);
	}
	
}

extern "C"
__global__ void update_stats_kernel()
{
	if(threadIdx.x == 0 && threadIdx.y == 0)
	{
		UInt64 totalIterationCount = *totalIterCountMem;
		
		UInt64 totalDotCount = 0;
		float peakDensity = 0;
		for(int row = 0; row < iterBlockCount; row++)
		{
			for(int col = 0; col < IterBlockSize; col++)
			{
				totalDotCount += dotCountBuffer[row][col];
				peakDensity = fmax(peakDensity, peakDensityBuffer[row][col]);
			}
		}
		float totalSubPixels = (float)(xRes*yRes*SubPixelCount);
		float density = (float)totalDotCount / totalSubPixels;
		
		float invPixArea = fabsf(vpsTransform.xa.x*vpsTransform.ya.y - vpsTransform.xa.y*vpsTransform.ya.x);
		//float scaleConstant = totalSubPixels / (float)totalDotCount;
		float scaleConstant = Tone_C2*(invPixArea*(float)SubPixelCount)/(float)totalIterationCount;
		
		*totalDotCountMem = totalDotCount;
		*densityMem = fmax(density,Epsilon);
		*peakDensityMem = fmax(peakDensity,Epsilon);
		*scaleConstantMem = fmax(scaleConstant,Epsilon);
	}
}

extern "C"
__device__ float4 tonemap(float4 rawPix, float scaleConstant)
{
	float z, gammaFactor;
	float4 logPix;
	float4 result;        //the tonemapped pixel
	
	float ka = Tone_C1 * brightness * __log10f(1.0f+rawPix.w*scaleConstant) / rawPix.w;
		
	logPix.x = rawPix.x*ka;
	logPix.y = rawPix.y*ka;
	logPix.z = rawPix.z*ka;
	logPix.w = rawPix.w*ka;
	
	z = __powf(logPix.w,invGamma);
	gammaFactor = z / logPix.w;
	
	result.x = __saturatef(lerp(__powf(logPix.x,invGamma), gammaFactor*logPix.x, vibrancy));
	result.y = __saturatef(lerp(__powf(logPix.y,invGamma), gammaFactor*logPix.y, vibrancy));
	result.z = __saturatef(lerp(__powf(logPix.z,invGamma), gammaFactor*logPix.z, vibrancy));
	result.w = __saturatef(z);
	
	return result;
}

extern "C"
__global__ void update_output_kernel(DeviceBuffer2D<uint> glOutputBuffer)
{
	//__shared__ float density;
	//__shared__ float peak_density;
	__shared__ float scaleConstant;
	
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x >= xRes || y >= yRes) return;
	
	
	if(threadIdx.x == 0 && threadIdx.y == 0)
	{
		//density = *densityMem;
		//peak_density = *peakDensityMem;
		scaleConstant = *scaleConstantMem;
	}
	__syncthreads();
	
	uint4 iPix;
	float4 pix;
	float4 acc;
	//float4 result;
	acc = tonemap( accumBuffer[y][4*x+0], scaleConstant);
		
	pix = tonemap( accumBuffer[y][4*x+1], scaleConstant);
	acc.x += pix.x; acc.y += pix.y; acc.z += pix.z; acc.w += pix.w;
	
	pix = tonemap( accumBuffer[y][4*x+2], scaleConstant);
	acc.x += pix.x; acc.y += pix.y; acc.z += pix.z; acc.w += pix.w;
	
	pix = tonemap( accumBuffer[y][4*x+3], scaleConstant);
	acc.x += pix.x; acc.y += pix.y; acc.z += pix.z; acc.w += pix.w;
	
	acc.x /= 4.0f;
	acc.y /= 4.0f;
	acc.z /= 4.0f;
	acc.w /= 4.0f;
	
	//result.x = lerp(bgColor.x, acc.x, acc.w);
	//result.y = lerp(bgColor.y, acc.y, acc.w);
	//result.z = lerp(bgColor.z, acc.z, acc.w);
	//result.w = 1.0f;
	
	iPix.x = (uint)(255.0f*acc.x);
	iPix.y = (uint)(255.0f*acc.y);
	iPix.z = (uint)(255.0f*acc.z);
	iPix.w = (uint)(255.0f*acc.w);
	
	glOutputBuffer[y][x] = iPix.w << 24 | iPix.z << 16 | iPix.y << 8 | iPix.x;
}
